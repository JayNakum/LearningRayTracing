﻿#include "hip/hip_runtime.h"


#include <iostream>
#include <time.h>

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const* const func, const char* const file, int const line)
{
    if (result)
    {
        std::cerr << "CUDA error [" << static_cast<unsigned int>(result) << "]: " <<
            file << "(" << line << ") '" << func << "' \n";

        hipDeviceReset();
        exit(99);
    }
}

__global__ void render(float* fb, int max_x, int max_y)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if ((i >= max_x) || (j >= max_y)) return;

    int pixel_index = (3*j*max_x) + (3*i);
    fb[pixel_index + 0] = float(i) / max_x;
    fb[pixel_index + 1] = float(j) / max_y;
    fb[pixel_index + 2] = 0.2f;
}

int main()
{
    int nx = 1200;
    int ny = 600;

    int num_pixels = nx * ny;
    size_t fb_size = 3 * num_pixels * sizeof(float);

    // allocate FB
    float* fb = nullptr;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    int tx = 8;
    int ty = 8;

    dim3 blocks(nx/tx+1, ny/ty+1);
    dim3 threads(tx, ty);

    clock_t start, stop;
    start = clock();

    render<<<blocks, threads>>>(fb, nx, ny);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    stop = clock();
    double time = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "Completed in " << time << " seconds." << std::endl;

    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny - 1; j >= 0; j--)
    {
        for (int i = 0; i < nx; i++)
        {
            int pixel_index = (3 * j * nx) + (3 * i);
            float r = fb[pixel_index + 0];
            float g = fb[pixel_index + 1];
            float b = fb[pixel_index + 2];
            int ir = int(255.99 * r);
            int ig = int(255.99 * g);
            int ib = int(255.99 * b);
            std::cout << ir << " " << ig << " " << ib << std::endl;
        }
    }

    checkCudaErrors(hipFree(fb));
}
